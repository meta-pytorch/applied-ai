#include "hip/hip_runtime.h"
#include "stochastic_rounding.hpp"
#include <cmath>

__device__ __forceinline__ PhiloxGenerator::PhiloxGenerator() :
    key(make_uint2(0, 0)),
    counter(make_uint4(0, 0, 0, 0)) {}

__device__ __forceinline__ void PhiloxGenerator::init(uint64_t seed, uint32_t thread_id) {
    key.x = static_cast<uint32_t>(seed);
    key.y = static_cast<uint32_t>(seed >> 32);
    counter = make_uint4(thread_id, 0, 0, 0);
    __threadfence_block();
}

__device__ __forceinline__ uint2 PhiloxGenerator::mulhilo(const unsigned int a, const unsigned int b) {
    uint2 result;
    unsigned long long prod;
    asm("mul.wide.u32 %0, %1, %2;" : "=l"(prod) : "r"(a), "r"(b));
    result.x = static_cast<unsigned int>(prod);
    result.y = static_cast<unsigned int>(prod >> 32);
    return result;
}

__device__ __forceinline__ uint4 PhiloxGenerator::round(uint4 ctr, uint2 key) {
    const uint2 mul0 = mulhilo(philox::M0, ctr.x);
    const uint2 mul1 = mulhilo(philox::M1, ctr.z);

    return make_uint4(
        mul1.y ^ ctr.y ^ key.x,
        mul1.x,
        mul0.y ^ ctr.w ^ key.y,
        mul0.x
    );
}

__device__ __forceinline__ uint4 PhiloxGenerator::next() {
    uint4 ctr = counter;
    uint2 k = key;

    #pragma unroll
    for (int i = 0; i < philox::ROUNDS; ++i) {
        ctr = round(ctr, k);
        k.x += philox::W32_0;
        k.y += philox::W32_1;
    }

    counter.x += 4;
    return ctr;
}

// BF16 stochastic rounding - 16 bits total, with 7 bits mantissa
__device__ __forceinline__ __hip_bfloat16 float_to_bf16_stochastic(const float value, const uint32_t rand) {
    // Handle special cases first
    if (!isfinite(value)) {
        return __float2bfloat16(value);
    }

    // Extract bits from float
    const uint32_t val_bits = __float_as_uint(value);

    // For BF16, we keep top 16 bits and use next 8 bits for rounding
    const uint32_t truncated = val_bits & 0xFFFF0000u;  // Top 16 bits
    const uint32_t rounding_bits = (val_bits >> 16) & 0xFF;  // Next 8 bits

    // Round up if random value is less than truncated bits
    const uint32_t random_bits = (rand & 0xFF);
    const uint32_t rounded = truncated + (random_bits < rounding_bits ? 0x10000u : 0);

    return __float2bfloat16(__uint_as_float(rounded));
}

// FP16 stochastic rounding - 16 bits total, with 10 bits mantissa
__device__ __forceinline__ __half float_to_fp16_stochastic(const float value, const uint32_t rand) {
    // Handle special cases first
    if (!isfinite(value)) {
        return __float2half(value);
    }

    const uint32_t val_bits = __float_as_uint(value);
    const uint32_t sign = val_bits & 0x80000000u;  // Extract sign bit
    const uint32_t exp = (val_bits >> 23) & 0xFFu;  // Extract exponent
    const uint32_t mant = val_bits & 0x7FFFFFu;     // Extract mantissa

    // Handle subnormals and exponent adjustment
    if (exp == 0) {
        // Input is subnormal or zero
        return __float2half(value);
    }

    // Adjust exponent bias from FP32 (127) to FP16 (15)
    const int new_exp = exp - 127 + 15;

    if (new_exp < 0) {
        // Result would be subnormal in FP16
        return __float2half(value);  // Let CUDA handle subnormal conversion
    }

    if (new_exp > 31) {
    // Would overflow FP16's exponent range
    return __float2half(sign ? -INFINITY : INFINITY);
}

    // FP16 mantissa is 10 bits, so we need to round from 23 to 10 bits
    const uint32_t mant_rounding_bits = mant & 0x1FFFu;  // Bottom 13 bits
    const uint32_t mant_msb = mant >> 13;               // Top 10 bits

    // Generate probability from the truncated bits
    const uint32_t random = rand & 0x1FFFu;  // Use 13 bits of randomness

    // Round up if random value is less than truncated bits
    const uint32_t round_up = (random < mant_rounding_bits) ? 1u : 0u;

    // Combine the pieces into FP16's bit pattern
    // [15] sign bit
    // [14:10] exponent (5 bits)
    // [9:0] mantissa (10 bits)
    const uint16_t h_bits = ((sign >> 16) & 0x8000u) |                 // Sign bit
                           ((new_exp & 0x1Fu) << 10) |        // Exponent
                           ((mant_msb + round_up) & 0x3FFu);  // Mantissa with rounding

    __half result;
    __half_raw* raw_ptr = reinterpret_cast<__half_raw*>(&result);
    raw_ptr->x = h_bits;
    return result;
}

__device__ __forceinline__ void float4_to_bf16_stochastic(
    const float4& values,
    uint4& rand_vals,
    __hip_bfloat16* output) {

    float vals[4] = {values.x, values.y, values.z, values.w};
    uint32_t rands[4] = {rand_vals.x, rand_vals.y, rand_vals.z, rand_vals.w};

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        output[i] = float_to_bf16_stochastic(vals[i], rands[i]);
    }
}

__device__ __forceinline__ void float4_to_fp16_stochastic(
    const float4& values,
    uint4& rand_vals,
    __half* output) {

    float vals[4] = {values.x, values.y, values.z, values.w};
    uint32_t rands[4] = {rand_vals.x, rand_vals.y, rand_vals.z, rand_vals.w};

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        output[i] = float_to_fp16_stochastic(vals[i], rands[i]);
    }
}

__global__ void stochastic_round_bf16(
    float *__restrict__ input,
    __hip_bfloat16 *__restrict__ output,
    int size,
    uint64_t seed) {

    PhiloxGenerator rng;
    rng.init(seed, blockIdx.x * blockDim.x + threadIdx.x);

    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    int stride = blockDim.x * gridDim.x * 4;

    float4 values;
    __hip_bfloat16 local_output[4];

    for (; idx <= size - 4; idx += stride) {
        values = *reinterpret_cast<float4*>(&input[idx]);
        uint4 rand = rng.next();
        float4_to_bf16_stochastic(values, rand, local_output);

        for (int j = 0; j < 4; j++) {
            output[idx + j] = local_output[j];
        }
    }

    if (idx < size) {
        float remaining_values[4] = {0.0f, 0.0f, 0.0f, 0.0f};
        int remainder = size - idx;

        for (int j = 0; j < remainder; j++) {
            remaining_values[j] = input[idx + j];
        }

        values.x = remaining_values[0];
        values.y = remaining_values[1];
        values.z = remaining_values[2];
        values.w = remaining_values[3];

        uint4 rand = rng.next();
        float4_to_bf16_stochastic(values, rand, local_output);

        for (int j = 0; j < remainder; j++) {
            output[idx + j] = local_output[j];
        }
    }
}

__global__ void stochastic_round_fp16(
    float *__restrict__ input,
    __half *__restrict__ output,
    int size,
    uint64_t seed) {

    PhiloxGenerator rng;
    rng.init(seed, blockIdx.x * blockDim.x + threadIdx.x);

    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    int stride = blockDim.x * gridDim.x * 4;

    float4 values;
    __half local_output[4];

    for (; idx <= size - 4; idx += stride) {
        values = *reinterpret_cast<float4*>(&input[idx]);
        uint4 rand = rng.next();
        float4_to_fp16_stochastic(values, rand, local_output);

        for (int j = 0; j < 4; j++) {
            output[idx + j] = local_output[j];
        }
    }

    if (idx < size) {
        float remaining_values[4] = {0.0f, 0.0f, 0.0f, 0.0f};
        int remainder = size - idx;

        for (int j = 0; j < remainder; j++) {
            remaining_values[j] = input[idx + j];
        }

        values.x = remaining_values[0];
        values.y = remaining_values[1];
        values.z = remaining_values[2];
        values.w = remaining_values[3];

        uint4 rand = rng.next();
        float4_to_fp16_stochastic(values, rand, local_output);

        for (int j = 0; j < remainder; j++) {
            output[idx + j] = local_output[j];
        }
    }
}
